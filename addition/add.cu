
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ 
void add(int *a, int *b, int *c)
{
	 *c = *a + *b;        // c[0] = a[0] + b[0];
}

int main(void) 
{
	int a = 12, b = 13, c;        // host copies
	int *d_a, *d_b, *d_c;		      // device copies

	// allocate space for device copies of a,b,c
	hipMalloc((void **) &d_a, sizeof(int));
	hipMalloc((void **) &d_b, sizeof(int));
	hipMalloc((void **) &d_c, sizeof(int));

	// copy a,b from host to the device
	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

	// launch add() kernel on GPU
	add<<<1,1>>>(d_a,d_b,d_c);

	// copy result back to host
	hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

	printf("%d\n", c);

	// cleanup
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
