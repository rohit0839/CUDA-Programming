
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__
void print_from_gpu()
{
	printf("Hello World! from device (GPU)\n");
}

int main(void) 
{
	printf("Hello World from host (CPU)\n");
	print_from_gpu<<<1,1>>>();
	return 0;
}

